#include <stdio.h>
#include <hip/hip_runtime.h>
#define	CUDASAFECALL(cmd, desc)	{								\
			if((res = (cmd)) != hipSuccess){					\
				printf("ERROR: %s [errno=%d]\n", desc, res);	\
				return 1;										\
			}													\
		}

int main(){
	int i, res, count;
	int major, minor;
	char name[100];
	hipDevice_t dev;

	CUDASAFECALL(hipInit(0), "Init CUDA Driver API");
	CUDASAFECALL(hipGetDeviceCount(&count), "Get number of GPUs w/ Compute Capability >= 1.0");
	for(i=0;i<count;i++){
		CUDASAFECALL(hipDeviceGet(&dev, 0), "Get device handle");
		CUDASAFECALL(hipDeviceGetName(name, 100, dev), "Get device name");
		CUDASAFECALL(hipDeviceComputeCapability(&major, &minor, dev), "Get device compute capability");
		printf("GPU#%d: %s\n", i, name);
		printf("Device Compute Cabability: %d.%d\n", major, minor, dev);
	}

	return 0;
}
